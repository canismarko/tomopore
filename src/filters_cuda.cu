#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "config.h"
#include "filters.h"
#include "filters_cuda.h"


// Local (cuda-only) function declarations
__global__ void tp_apply_kernel_cuda_kernel
(Matrix3D *working_buffer, Matrix3D *kernel, Matrix2D *pore_slice_buffer,
 enum operation op, DIM row_start, DIM row_end, DIM islc);
__device__ float tp_apply_kernel(Matrix3D *subvolume, Matrix3D *kernel, DIM islc, DIM irow, DIM icol, enum operation op);


// Function definitions
/* Take a buffer of data, and apply the kernel to each row/column pixel for the given slice index
 *islc*, *irow*, *icol* give the current position in the *subvolume* buffer */
__device__ float tp_apply_kernel(Matrix3D *subvolume, Matrix3D *kernel, DIM islc, DIM irow, DIM icol, enum operation op)
// i, j, k -> indices of subvolume
// l, m, n -> indices of kernel
// dL, dM, dN -> reach of the kernel, so for a 3x3x3 kernel, each is (3-1)/2 = 1
// Calculate some values to relate between the kernel and the subvolume
{
  DIM dL = (kernel->nslices - 1) / 2;
  DIM dM = (kernel->nrows - 1) / 2;
  DIM dN = (kernel->ncolumns - 1) / 2;
  int64_t i, j, k;
  DDIM subvolume_slice_idx = 0;
  DDIM subvolume_idx = 0;
  DDIM kernel_idx = 0;
  DTYPE kernel_val, volume_val;
  int is_in_bounds;
  char is_first_round = TRUE;
  double running_total = 0;
  char replace_value = FALSE;
  // Iterate over the kernel dimensions, then apply them to the main arr
  // i, j, k are in the buffer space
  // l, m, n are in the kernel space
  for (DIM l=0; l < kernel->nslices; l++) {
    i = islc + (l - dL);
    // Check if this is a valid slice in the subvolume
    is_in_bounds = (i >= 0) && (i < subvolume->nslices);
    if (!is_in_bounds) continue;
    // Keep track of where we are in the subvolume
    subvolume_slice_idx = i * subvolume->nrows * subvolume->ncolumns;
    for (DIM m=0; m < kernel->nrows; m++) {
      j = irow + (m - dM);
      // Check if this is a valid row in the subvolume
      is_in_bounds = (j >= 0) && (j < subvolume->nrows);
      if (!is_in_bounds) continue;
      // Keep track of where we are in the subvolume
      subvolume_idx = subvolume_slice_idx + j * subvolume->ncolumns;
      for (DIM n=0; n < kernel->ncolumns; n++) {
	// Calculate relative coordinates in the arr matrix
	k = icol + (n - dN);
	// Check if this is a valid column in the subvolume
	is_in_bounds = (k >= 0) && (k < subvolume->ncolumns);
	if (!is_in_bounds) continue;
	// Retrieve the values from arrays and perform the actual
	// operation
	if (is_in_bounds) {
	  volume_val = subvolume->arr[subvolume_idx + k];
	  kernel_val = kernel->arr[kernel_idx];
	  // Apply the actual kernel filter function
	  if (kernel_val > 0) {
	    // Set the beginning value if one hasn't been set yet
	    if (is_first_round) {
	      running_total = volume_val;
	      is_first_round = FALSE;
	    }
	    // Save this value as the new minimum/maximum if it's bigger than the old one
	    replace_value = (volume_val < running_total) && (op == Min);
	    replace_value = replace_value || ((volume_val > running_total) && (op == Max));
	    if (replace_value) {
	      running_total = volume_val;
	    }
	  }	  
	}
	kernel_idx++;
      }
    }
  }
  return (float) running_total;
}


__global__ void tp_apply_kernel_cuda_kernel
(Matrix3D *working_buffer, Matrix3D *kernel, Matrix2D *pore_slice_buffer,
 enum operation op, DIM islc)
{
  DIM n_rows = working_buffer->nrows;
  DIM n_cols = working_buffer->ncolumns;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (DIM irow=index; irow < n_rows; irow+=stride) {
    for (DIM icol=0; icol < n_cols; icol++) {
      DDIM this_idx = irow * pore_slice_buffer->ncolumns + icol;
      pore_slice_buffer->arr[this_idx] = tp_apply_kernel(working_buffer,
							 kernel,
							 islc,
							 irow,
							 icol,
							 op);
    }
  }
    // DIM rows_per_thread = (DIM) ceil((double) n_rows / (double) config.n_threads);
    // DIM next_row = 0;
    // pthread_t tids[config.n_threads];
    // ThreadPayload *payload;
    // for (DIM tidx=0; tidx < config.n_threads; tidx++) {
    //   if (next_row < n_rows) {
    // 	payload = (ThreadPayload *) malloc(sizeof(ThreadPayload));
    // 	payload->row_start = next_row;
    // 	next_row += rows_per_thread;
    // 	payload->row_end = min_d(next_row, n_rows);
    // 	payload->n_cols = n_cols;
    // 	payload->new_islc = new_islc;
    // 	pthread_create(&tids[tidx], NULL, tp_apply_kernel_thread, payload);
    //   } else {
    // 	tids[tidx] = 0;
    //   }
    // }
    // // Wait for threads to finish
    // for (pthread_t tidx=0; tidx < config.n_threads; tidx++) {
    //   if (tids[tidx] > 0) {
    // 	pthread_join(tids[tidx], NULL);
    //   }
    // }
}



char tp_apply_kernel_cuda
(Matrix3D *working_buffer, Matrix3D *kernel, Matrix2D *pore_slice_buffer, enum operation op, DIM islc)
{
  // printf("Before: %f\n", working_buffer->arr[0]);
  hipProfilerStart();
  tp_apply_kernel_cuda_kernel<<<1, 512>>>(working_buffer, kernel, pore_slice_buffer, op, islc);
  hipDeviceSynchronize();
  hipProfilerStop();
  // printf("After: %f\n", working_buffer->arr[0]);
  return 0;
}
