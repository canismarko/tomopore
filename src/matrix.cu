#include <stdio.h>

#include "matrix.h"

// Take a volume and return a flattened index for a given slice, row and column
DDIM tp_indices(Matrix3D *vol, DIM islice, DIM irow, DIM icolumn) {
  return islice * vol->nrows * vol->ncolumns + irow * vol->ncolumns + icolumn;
}


// Take a slice and return a flattened index for a given row and column
__host__ __device__ DDIM tp_indices2d(Matrix2D *vol, DIM irow, DIM icolumn) {
  return irow * vol->ncolumns + icolumn;
}

Matrix3D *tp_matrixmalloc(DIM n_slices, DIM n_rows, DIM n_columns) {
  // Allocated memory for the 3D matrix
  // Matrix3D *new_matrix = (Matrix3D *) hipMallocManaged(sizeof(Matrix3D) + n_slices * n_rows * n_columns * sizeof(DTYPE));
  Matrix3D *new_matrix;
  hipMallocManaged(&new_matrix, sizeof(Matrix3D) + n_slices * n_rows * n_columns * sizeof(DTYPE));
  if (new_matrix == NULL) {
    fprintf(stderr, "Unable to allocate memory for (%lu, %lu, %lu) array.", n_slices, n_rows, n_columns);
  } 
  // Store the size of the array
  new_matrix->nslices = n_slices;
  new_matrix->nrows = n_rows;
  new_matrix->ncolumns = n_columns;
  return new_matrix;
}


Matrix2D *tp_matrixmalloc2d(DIM n_rows, DIM n_columns) {
  // Allocated memory for the 2D matrix
  Matrix2D *new_matrix;
  hipMallocManaged(&new_matrix, sizeof(Matrix2D) + n_rows * n_columns * sizeof(DTYPE));
  if (new_matrix == NULL) {
    fprintf(stderr, "Unable to allocate memory for (%lu, %lu) array.", n_rows, n_columns);
  } 
  // Store the size of the array
  new_matrix->nrows = n_rows;
  new_matrix->ncolumns = n_columns;
  return new_matrix;
}
